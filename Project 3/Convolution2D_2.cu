/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	5

void checkCudaErrors(hipError_t error){
	if(error != hipSuccess) {
		printf("\033[0;31mCUDA Error: %s in %s, line %d\033[0;37m\n", hipGetErrorString(error), __FILE__, __LINE__);
	}
}



////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, 
                       int imageW, int imageH, int filterR) {
  	int k;
                      
	float sum = 0;

	for (k = -filterR; k <= filterR; k++) {
		int d = threadIdx.x + k;

		if (d >= 0 && d < imageW)
			sum += d_Src[threadIdx.y  * imageW + d] * d_Filter[filterR - k];    

        d_Dst[threadIdx.y * imageW + threadIdx.x] = sum;
 	}
        
}

__global__ void convolutionColumnGPU(float *d_Dst, float *d_Src, float *d_Filter,
    			   int imageW, int imageH, int filterR) {

	int k;
  
	float sum = 0;

	for (k = -filterR; k <= filterR; k++) {
		int d = threadIdx.y + k;

        if (d >= 0 && d < imageH)
			sum += d_Src[d * imageW + threadIdx.x] * d_Filter[filterR - k];
 
        d_Dst[threadIdx.y * imageW + threadIdx.x] = sum;
	}
    
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU,
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;


    int imageW;
    int imageH;
    unsigned int i;

	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
    h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
    h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
    h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
		
	if( !h_Filter || !h_Input|| !h_Buffer || !h_OutputCPU || !h_OutputGPU ){
		printf("Error while allocating host memory.\n");
		exit(1);
    }

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (float)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
    }
    
    printf("Allocating and initializing device arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    hipMalloc( (void**)&d_Filter, FILTER_LENGTH * sizeof(float));
    //checkCudaErrors(cudaGetLastError());
    hipMalloc( (void**)&d_Input, imageW * imageH * sizeof(float));
    //checkCudaErrors(cudaGetLastError());
    hipMalloc( (void**)&d_Buffer, imageW * imageH *sizeof(float));
    //checkCudaErrors(cudaGetLastError());
    hipMalloc( (void**)&d_OutputGPU,imageW * imageH * sizeof(float));
    //checkCudaErrors(cudaGetLastError());
    
    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(float), hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(float), hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());
    

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

	printf("GPU computation...\n");
	
	dim3 grid_dim(1, 1);
    dim3 block_dim(imageW, imageH);
    
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
    
    convolutionRowGPU<<<grid_dim, block_dim>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
	convolutionColumnGPU<<<grid_dim, block_dim>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius);
 
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	
	hipMemcpy( h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(float) , hipMemcpyDeviceToHost);
	//cudaDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  

    for(i=0; i < imageW * imageH; i++){
//	printf("cpu :%lf , gpu: %lf \n",h_OutputCPU[i], h_OutputGPU[i]);
    	if( ABS(h_OutputGPU[i] - h_OutputCPU[i]) > accuracy) {
    		printf("\033[0;35mGpu output differs\033[0;37m\n");
    		break;
    	 }
    }


    // free all the allocated memory
    free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);
	free(h_OutputGPU);
	hipFree(d_OutputGPU);
	hipFree(d_Buffer);
	hipFree(d_Input);
	hipFree(d_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
