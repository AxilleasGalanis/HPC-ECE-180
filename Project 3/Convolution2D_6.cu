/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.000005

void checkCudaErrors(hipError_t error){
	if(error != hipSuccess) {
		printf("\033[0;31mCUDA Error: %s in %s, line %d\033[0;37m\n", hipGetErrorString(error), __FILE__, __LINE__);
	}
}



////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      double sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }
    
}

__global__ void convolutionRowGPU(double *d_Dst, double *d_Src, double *d_Filter, 
                       int imageW, int imageH, int filterR) {
  	int k;
                      
	double sum = 0;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x < imageW && y < imageH){ 
		for (k = -filterR; k <= filterR; k++) {
			int d = x + k;

			if (d >= 0 && d < imageW)
				sum += d_Src[y  * imageW + d] * d_Filter[filterR - k];    

		    d_Dst[y * imageW + x] = sum;
	 	}
	}   
}

__global__ void convolutionColumnGPU(double *d_Dst, double *d_Src, double *d_Filter,
    			   int imageW, int imageH, int filterR) {

	int k;
  
	double sum = 0;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if(x < imageW && y < imageH){ 
		for (k = -filterR; k <= filterR; k++) {
			int d = y + k;

		    if (d >= 0 && d < imageH)
				sum += d_Src[d * imageW + x] * d_Filter[filterR - k];
	 
		    d_Dst[y * imageW + x] = sum;
		}
	}
    
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    double
    *h_Filter,
    *h_Input,
    *h_Buffer,
    *h_OutputCPU,
    *h_OutputGPU,
    *d_Filter,
    *d_Input,
    *d_Buffer,
    *d_OutputGPU;


    int imageW;
    int imageH;
    unsigned int i;
	struct timespec tv1, tv2;
	hipEvent_t start, stop;
	
	printf("Enter filter radius : ");
	scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    scanf("%d", &imageW);
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (double *)malloc(FILTER_LENGTH * sizeof(double));
    h_Input     = (double *)malloc(imageW * imageH * sizeof(double));
    h_Buffer    = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputCPU = (double *)malloc(imageW * imageH * sizeof(double));
    h_OutputGPU = (double *)malloc(imageW * imageH * sizeof(double));
		
	if( !h_Filter || !h_Input|| !h_Buffer || !h_OutputCPU || !h_OutputGPU ){
		printf("Error while allocating host memory.\n");
		exit(1);
    }

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (double)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + (double)rand() / (double)RAND_MAX;
    }
    
    printf("Allocating and initializing device arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    hipMalloc( (void**)&d_Filter, FILTER_LENGTH * sizeof(double));
    //checkCudaErrors(cudaGetLastError());
    hipMalloc( (void**)&d_Input, imageW * imageH * sizeof(double));
    //checkCudaErrors(cudaGetLastError());
    hipMalloc( (void**)&d_Buffer, imageW * imageH *sizeof(double));
    //checkCudaErrors(cudaGetLastError());
    hipMalloc( (void**)&d_OutputGPU,imageW * imageH * sizeof(double));
    //checkCudaErrors(cudaGetLastError());
    
    hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(double), hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());
    hipMemcpy(d_Input, h_Input, imageW * imageH * sizeof(double), hipMemcpyHostToDevice);
    checkCudaErrors(hipGetLastError());
    

	hipEventCreate(&start);
    hipEventCreate(&stop);

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

	clock_gettime(CLOCK_MONOTONIC_RAW,&tv1);
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles
	clock_gettime(CLOCK_MONOTONIC_RAW,&tv2);

	printf("GPU computation...\n");
	
	hipEventRecord(start);
	
	dim3 block_dim(32, 32);
	dim3 grid_dim(ceil((double)imageW/32), ceil((double)imageH/32));
    
    
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
    
    convolutionRowGPU<<<grid_dim, block_dim>>>(d_Buffer, d_Input, d_Filter, imageW, imageH, filter_radius);
    hipDeviceSynchronize();
	convolutionColumnGPU<<<grid_dim, block_dim>>>(d_OutputGPU, d_Buffer, d_Filter, imageW, imageH, filter_radius);
 
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	
	hipMemcpy( h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(double) , hipMemcpyDeviceToHost);
	//cudaDeviceSynchronize();
	checkCudaErrors(hipGetLastError());
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
    float diff = 0;
    hipEventElapsedTime(&diff, start, stop);
    
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  

    for(i=0; i < imageW * imageH; i++){
//	printf("cpu :%lf , gpu: %lf \n",h_OutputCPU[i], h_OutputGPU[i]);
    	if( ABS(h_OutputGPU[i] - h_OutputCPU[i]) > accuracy) {
    		printf("\033[0;35mGpu output differs\033[0;37m\n");
    		break;
    	 }
    }

	printf("CPU time = %.10f seconds\n", (double) (tv2.tv_nsec - tv1.tv_nsec)/1000000000.0 
	+ (double) (tv2.tv_sec - tv1.tv_sec));
	
	 printf("GPU time = %.10f seconds\n", diff / 1000);
    // free all the allocated memory
    free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);
	free(h_OutputGPU);
	hipFree(d_OutputGPU);
	hipFree(d_Buffer);
	hipFree(d_Input);
	hipFree(d_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipDeviceReset();


    return 0;
}
